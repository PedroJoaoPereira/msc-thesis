#include "hip/hip_runtime.h"
#include "CUDA_Resample.h"

// global declaration of 2D float texture (visible for host and device code)
texture<uint8_t, hipTextureType2D, hipReadModeElementType> tex;

// Allocate image channels data buffers depending of the pixel format
void cudaAllocBuffers(uint8_t** &buffer, int* &bufferSize, int width, int height, int pixelFormat){
    // Allocate channel buffer size
    bufferSize = static_cast<int*>(malloc(3 * sizeof(int)));

    // Calculate once
    int wxh = width * height;
    int wxhDiv2 = wxh / 2;
    int wxhDiv4 = wxh / 4;

    // Calculate buffer sizes for each pixel format
    switch(pixelFormat){
    case AV_PIX_FMT_UYVY422:
        bufferSize[0] = wxh * 2;
        bufferSize[1] = 0;
        bufferSize[2] = 0;
        break;
    case AV_PIX_FMT_YUV422P:
        bufferSize[0] = wxh;
        bufferSize[1] = wxhDiv2;
        bufferSize[2] = wxhDiv2;
        break;
    case AV_PIX_FMT_YUV420P:
        bufferSize[0] = wxh;
        bufferSize[1] = wxhDiv4;
        bufferSize[2] = wxhDiv4;
        break;
    case AV_PIX_FMT_NV12:
        bufferSize[0] = wxh;
        bufferSize[1] = wxhDiv2;
        bufferSize[2] = 0;
        break;
    case AV_PIX_FMT_V210:
        bufferSize[0] = height * 128 * ((width + 47) / 48);
        bufferSize[1] = 0;
        bufferSize[2] = 0;
        break;
    case AV_PIX_FMT_YUV422PNORM:
        bufferSize[0] = wxh;
        bufferSize[1] = wxhDiv2;
        bufferSize[2] = wxhDiv2;
        break;
    }

    // Allocate buffer memory
    buffer = static_cast<uint8_t**>(malloc(3 * sizeof(uint8_t*)));

    // Allocate buffer in the GPU memory
    hipMalloc((void **) &buffer[0], bufferSize[0]);
    if(bufferSize[1] != 0)
        hipMalloc((void **) &buffer[1], bufferSize[1]);
    if(bufferSize[2] != 0)
        hipMalloc((void **) &buffer[2], bufferSize[2]);
}

// Free used GPU memory
void freeCudaMemory(uint8_t** &buffer){
    // Iterate each channel and free memory
    for(int i = 0; i < 3; i++)
        hipFree(buffer[i]);

    // Free host memory
    free(buffer);
}

// Copy data from host to device
void cudaCopyBuffersToGPU(uint8_t* srcBuffer[], uint8_t* gpuBuffer[], int* &bufferSize){
    // First channel
    hipMemcpy(gpuBuffer[0], srcBuffer[0], bufferSize[0], hipMemcpyHostToDevice);

    // Copy chroma channels if they exist
    if(bufferSize[1] != 0)
        hipMemcpy(gpuBuffer[1], srcBuffer[1], bufferSize[1], hipMemcpyHostToDevice);
    if(bufferSize[2] != 0)
        hipMemcpy(gpuBuffer[2], srcBuffer[2], bufferSize[2], hipMemcpyHostToDevice);
}

// Copy data from device to host
void cudaCopyBuffersFromGPU(uint8_t* targetBuffer[], uint8_t* gpuBuffer[], int* &bufferSize){
    // First channel
    hipMemcpy(targetBuffer[0], gpuBuffer[0], bufferSize[0], hipMemcpyDeviceToHost);

    // Copy chroma channels if they exist
    if(bufferSize[1] != 0)
        hipMemcpy(targetBuffer[1], gpuBuffer[1], bufferSize[1], hipMemcpyDeviceToHost);
    if(bufferSize[2] != 0)
        hipMemcpy(targetBuffer[2], gpuBuffer[2], bufferSize[2], hipMemcpyDeviceToHost);
}

// Calculate launch parameters of format conversion kernel
pair<dim3, dim3> calculateConversionLP(int width, int height, int srcPixelFormat, int dstPixelFormat){
    // Variable with result launch parameters
    pair<dim3, dim3> result;

    // Discover dimensions value depending of the conversion
    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_UYVY422)
        result.first = dim3(width * 2, height);
    else if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_YUV422P)
        result.first = dim3(width / 2, height);
    else if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_YUV420P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_NV12)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width * 2 / 12, height);

    else if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_UYVY422)
        result.first = dim3(width / 2, height);
    else if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_YUV422P)
        result.first = dim3(width / 2, height);
    else if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_YUV420P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_NV12)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width / 6, height);

    else if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_UYVY422)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_YUV422P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_YUV420P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_NV12)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width / 6, height / 2);

    else if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_UYVY422)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_YUV422P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_YUV420P)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_NV12)
        result.first = dim3(width / 2, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width / 6, height / 2);

    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_UYVY422)
        result.first = dim3(width / 6, height);
    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV422P)
        result.first = dim3(width / 6, height);
    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV420P)
        result.first = dim3(width / 6, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_NV12)
        result.first = dim3(width / 6, height / 2);
    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width / 3, height);
    else if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV422PNORM)
        result.first = dim3(width / 6, height);

    else if(srcPixelFormat == AV_PIX_FMT_YUV422PNORM && dstPixelFormat == AV_PIX_FMT_V210)
        result.first = dim3(width / 6, height);

    // Calculate thread size
    int hDivisor = greatestDivisor(result.first.x, 16);
    int vDivisor = greatestDivisor(result.first.y, 16);

    // Assign thread size
    result.second = dim3(hDivisor, vDivisor);

    // Calculate block size
    result.first.x /= hDivisor;
    result.first.y /= vDivisor;

    return result;
}

// Calculate launch parameters of resize kernel
pair<dim3, dim3> calculateResizeLP(int width, int height, int initDivisor){
    // Variable with result launch parameters
    pair<dim3, dim3> result;

    // Dimensions are always the same because only deal with planar formats
    result.first = dim3(width, height);

    // Calculate thread size
    int hDivisor = greatestDivisor(result.first.x, initDivisor);
    int vDivisor = greatestDivisor(result.first.y, initDivisor);

    // Assign thread size
    result.second = dim3(hDivisor, vDivisor);

    // Calculate block size
    result.first.x /= hDivisor;
    result.first.y /= vDivisor;

    return result;
}

// ------------------------------------------------------------------

// Convert the pixel format of the image
void cuda_omp_formatConversion(int width, int height, int srcPixelFormat, uint8_t* srcSlice[], int dstPixelFormat, uint8_t* dstSlice[]){
    #pragma region UYVY422
    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_UYVY422){
        // Used metrics
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;

        // Copy data
        memcpy(dstSlice[0], srcSlice[0], vStrideUYVY422 * hStrideUYVY422);

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_YUV422P){
        // Used metrics
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        // Iterate blocks of 1x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideUYVY422; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideUYVY422;
            auto dstB = dstSlice[0] + vIndex * hStrideYUV422P;
            auto dstU = dstSlice[1] + vIndex * hStrideYUV422P / 2;
            auto dstV = dstSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideUYVY422 / 4; hIndex++){
                *dstU++ = *srcB++; // U0
                *dstB++ = *srcB++; // Y0
                *dstV++ = *srcB++; // V0
                *dstB++ = *srcB++; // Y1
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_YUV420P){
        // Used metrics
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;

        // Iterate blocks of 2x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideUYVY422 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideUYVY422 * 2;
            auto srcBb = srcB + hStrideUYVY422;
            auto dstB = dstSlice[0] + vIndex * hStrideYUV420P * 2;
            auto dstBb = dstB + hStrideYUV420P;
            auto dstU = dstSlice[1] + vIndex * hStrideYUV420P / 2;
            auto dstV = dstSlice[2] + vIndex * hStrideYUV420P / 2;

            for(int hIndex = 0; hIndex < hStrideUYVY422 / 4; hIndex++){
                // Get above line
                uint8_t u0 = *srcB++; // U0
                uint8_t y0 = *srcB++; // Y0
                uint8_t v0 = *srcB++; // V0
                uint8_t y1 = *srcB++; // Y1

                                      // Get below line
                *srcBb++; // U0
                uint8_t y0b = *srcBb++; // Y0
                *srcBb++; // V0
                uint8_t y1b = *srcBb++; // Y1

                                        // Assign above luma values
                *dstB++ = y0;
                *dstB++ = y1;

                // Assign below luma values
                *dstBb++ = y0b;
                *dstBb++ = y1b;

                // Assigne chroma values
                *dstU++ = u0;
                *dstV++ = v0;
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_NV12){
        // Used metrics
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;
        int vStrideNV12 = height;
        int hStrideNV12 = width;

        // Iterate blocks of 2x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideUYVY422 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideUYVY422 * 2;
            auto srcBb = srcB + hStrideUYVY422;
            auto dstB = dstSlice[0] + vIndex * hStrideNV12 * 2;
            auto dstBb = dstB + hStrideNV12;
            auto dstC = dstSlice[1] + vIndex * hStrideNV12;

            for(int hIndex = 0; hIndex < hStrideUYVY422 / 4; hIndex++){
                // Get above line
                uint8_t u0 = *srcB++; // U0
                uint8_t y0 = *srcB++; // Y0
                uint8_t v0 = *srcB++; // V0
                uint8_t y1 = *srcB++; // Y1

                                      // Get below line
                *srcBb++; // U0
                uint8_t y0b = *srcBb++; // Y0
                *srcBb++; // V0
                uint8_t y1b = *srcBb++; // Y1

                                        // Assign above luma values
                *dstB++ = y0;
                *dstB++ = y1;

                // Assign below luma values
                *dstBb++ = y0b;
                *dstBb++ = y1b;

                // Assigne chroma values
                *dstC++ = u0;
                *dstC++ = v0;
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_UYVY422 && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Iterate blocks of 1x12 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideUYVY422; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideUYVY422;
            auto dstB = reinterpret_cast<uint32_t*>(dstSlice[0]) + vIndex * hStrideV210;

            for(int hIndex = 0; hIndex < hStrideUYVY422 / 12; hIndex++){
                // Get components from source
                auto u0 = *srcB++ << 2U; // U0
                auto y0 = *srcB++ << 2U; // Y0
                auto v0 = *srcB++ << 2U; // V0
                auto y1 = *srcB++ << 2U; // Y1

                auto u1 = *srcB++ << 2U; // U1
                auto y2 = *srcB++ << 2U; // Y2
                auto v1 = *srcB++ << 2U; // V1
                auto y3 = *srcB++ << 2U; // Y3

                auto u2 = *srcB++ << 2U; // U2
                auto y4 = *srcB++ << 2U; // Y4
                auto v2 = *srcB++ << 2U; // V2
                auto y5 = *srcB++ << 2U; // Y5

                                         // Assign value
                *dstB++ = (v0 << 20U) | (y0 << 10U) | u0;
                *dstB++ = (y2 << 20U) | (u1 << 10U) | y1;
                *dstB++ = (u2 << 20U) | (y3 << 10U) | v1;
                *dstB++ = (y5 << 20U) | (v2 << 10U) | y4;
            }
        }

        return;
    }
    #pragma endregion

    #pragma region YUV422P
    if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_UYVY422){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;

        // Iterate blocks of 1x2 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideYUV422P; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideYUV422P;
            auto dstB = dstSlice[0] + vIndex * hStrideUYVY422;
            auto srcU = srcSlice[1] + vIndex * hStrideYUV422P / 2;
            auto srcV = srcSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideYUV422P / 2; hIndex++){
                *dstB++ = *srcU++; // U0
                *dstB++ = *srcB++; // Y0
                *dstB++ = *srcV++; // V0
                *dstB++ = *srcB++; // Y1
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_YUV422P){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        // Copy data
        memcpy(dstSlice[0], srcSlice[0], vStrideYUV422P * hStrideYUV422P);
        memcpy(dstSlice[1], srcSlice[1], vStrideYUV422P * hStrideYUV422P / 2);
        memcpy(dstSlice[2], srcSlice[2], vStrideYUV422P * hStrideYUV422P / 2);

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_YUV420P){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;

        int hStrideYUV422PChroma = hStrideYUV422P / 2;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideYUV422P * hStrideYUV422P);

            // Iterate blocks of 2x1 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideYUV422P / 2; vIndex++){
                // Discover buffer pointers
                auto srcU = srcSlice[1] + vIndex * 2 * hStrideYUV422PChroma;
                auto srcV = srcSlice[2] + vIndex * 2 * hStrideYUV422PChroma;
                auto srcUb = srcU + hStrideYUV422PChroma;
                auto srcVb = srcV + hStrideYUV422PChroma;
                auto dstU = dstSlice[1] + vIndex * hStrideYUV420P / 2;
                auto dstV = dstSlice[2] + vIndex * hStrideYUV420P / 2;

                for(int hIndex = 0; hIndex < hStrideYUV422PChroma; hIndex++){
                    // Get above chroma values
                    uint8_t u = *srcU++; // U0
                    uint8_t v = *srcV++; // V0

                                         // Get below chroma values
                    uint8_t ub = *srcUb++; // U1
                    uint8_t vb = *srcVb++; // V1

                                           // Assign values
                    *dstU++ = uint8_t(roundFast((static_cast<double>(u) + static_cast<double>(ub)) / 2.));
                    *dstV++ = uint8_t(roundFast((static_cast<double>(v) + static_cast<double>(vb)) / 2.));
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_NV12){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;
        int vStrideNV12 = height;
        int hStrideNV12 = width;

        int hStrideYUV422PChroma = hStrideYUV422P / 2;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideYUV422P * hStrideYUV422P);

            // Iterate blocks of 2x1 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideYUV422P / 2; vIndex++){
                // Discover buffer pointers
                auto srcU = srcSlice[1] + vIndex * 2 * hStrideYUV422PChroma;
                auto srcV = srcSlice[2] + vIndex * 2 * hStrideYUV422PChroma;
                auto srcUb = srcU + hStrideYUV422PChroma;
                auto srcVb = srcV + hStrideYUV422PChroma;
                auto dstC = dstSlice[1] + vIndex * hStrideNV12;

                for(int hIndex = 0; hIndex < hStrideYUV422PChroma; hIndex++){
                    // Get above chroma values
                    uint8_t u = *srcU++; // U0
                    uint8_t v = *srcV++; // V0

                                         // Get below chroma values
                    uint8_t ub = *srcUb++; // U1
                    uint8_t vb = *srcVb++; // V1

                                           // Assign values
                    *dstC++ = uint8_t(roundFast((static_cast<double>(u) + static_cast<double>(ub)) / 2.));
                    *dstC++ = uint8_t(roundFast((static_cast<double>(v) + static_cast<double>(vb)) / 2.));
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV422P && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Iterate blocks of 1x6 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideYUV422P; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideYUV422P;
            auto dstB = reinterpret_cast<uint32_t*>(dstSlice[0]) + vIndex * hStrideV210;
            auto srcU = srcSlice[1] + vIndex * hStrideYUV422P / 2;
            auto srcV = srcSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideYUV422P / 6; hIndex++){
                // Get components from source
                auto u0 = *srcU++ << 2U; // U0
                auto y0 = *srcB++ << 2U; // Y0
                auto v0 = *srcV++ << 2U; // V0
                auto y1 = *srcB++ << 2U; // Y1

                auto u1 = *srcU++ << 2U; // U1
                auto y2 = *srcB++ << 2U; // Y2
                auto v1 = *srcV++ << 2U; // V1
                auto y3 = *srcB++ << 2U; // Y3

                auto u2 = *srcU++ << 2U; // U2
                auto y4 = *srcB++ << 2U; // Y4
                auto v2 = *srcV++ << 2U; // V2
                auto y5 = *srcB++ << 2U; // Y5

                                         // Assign value
                *dstB++ = (v0 << 20U) | (y0 << 10U) | u0;
                *dstB++ = (y2 << 20U) | (u1 << 10U) | y1;
                *dstB++ = (u2 << 20U) | (y3 << 10U) | v1;
                *dstB++ = (y5 << 20U) | (v2 << 10U) | y4;
            }
        }

        return;
    }
    #pragma endregion

    #pragma region YUV420P
    if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_UYVY422){
        // Used metrics
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;

        // Iterate blocks of 2x2 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideYUV420P / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * 2 * hStrideYUV420P;
            auto srcBb = srcB + hStrideYUV420P;
            auto dstB = dstSlice[0] + vIndex * 2 * hStrideUYVY422;
            auto dstBb = dstB + hStrideUYVY422;
            auto srcU = srcSlice[1] + vIndex * hStrideYUV420P / 2;
            auto srcV = srcSlice[2] + vIndex * hStrideYUV420P / 2;

            for(int hIndex = 0; hIndex < hStrideYUV420P / 2; hIndex++){
                // Get chroma values
                uint8_t u = *srcU++; // U
                uint8_t v = *srcV++; // V

                                     // Assign above line values
                *dstB++ = u; // U0
                *dstB++ = *srcB++; // Y0
                *dstB++ = v; // V0
                *dstB++ = *srcB++; // Y1

                                   // Assign below line values
                *dstBb++ = u; // U0
                *dstBb++ = *srcBb++; // Y0
                *dstBb++ = v; // V0
                *dstBb++ = *srcBb++; // Y1
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_YUV422P){
        // Used metrics
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        int hStrideYUV422PChroma = hStrideYUV422P / 2;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideYUV420P * hStrideYUV420P);

            // Iterate blocks of 2x2 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideYUV420P / 2; vIndex++){
                // Discover buffer pointers
                auto srcU = srcSlice[1] + vIndex * hStrideYUV420P / 2;
                auto srcV = srcSlice[2] + vIndex * hStrideYUV420P / 2;
                auto dstU = dstSlice[1] + vIndex * 2 * hStrideYUV422PChroma;
                auto dstV = dstSlice[2] + vIndex * 2 * hStrideYUV422PChroma;
                auto dstUb = dstU + hStrideYUV422PChroma;
                auto dstVb = dstV + hStrideYUV422PChroma;

                for(int hIndex = 0; hIndex < hStrideYUV420P / 2; hIndex++){
                    // Get chroma values
                    uint8_t u = *srcU++; // U
                    uint8_t v = *srcV++; // V

                                         // Assign values dupicated
                    *dstU++ = u;
                    *dstV++ = v;

                    *dstUb++ = u;
                    *dstVb++ = v;
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_YUV420P){
        // Used metrics
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;

        // Copy data
        memcpy(dstSlice[0], srcSlice[0], vStrideYUV420P * hStrideYUV420P);
        memcpy(dstSlice[1], srcSlice[1], vStrideYUV420P * hStrideYUV420P / 4);
        memcpy(dstSlice[2], srcSlice[2], vStrideYUV420P * hStrideYUV420P / 4);

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_NV12){
        // Used metrics
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;
        int vStrideNV12 = height;
        int hStrideNV12 = width;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideYUV420P * hStrideYUV420P);

            // Iterate blocks of 2x2 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideYUV420P / 2; vIndex++){
                // Discover buffer pointers
                auto srcU = srcSlice[1] + vIndex * hStrideYUV420P / 2;
                auto srcV = srcSlice[2] + vIndex * hStrideYUV420P / 2;
                auto dstC = dstSlice[1] + vIndex * hStrideNV12;

                for(int hIndex = 0; hIndex < hStrideYUV420P / 2; hIndex++){
                    *dstC++ = *srcU++; // U
                    *dstC++ = *srcV++; // V
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_YUV420P && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Iterate blocks of 2x2 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideYUV420P / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * 2 * hStrideYUV420P;
            auto srcBb = srcB + hStrideYUV420P;
            auto dstB = reinterpret_cast<uint32_t*>(dstSlice[0]) + vIndex * 2 * hStrideV210;
            auto dstBb = dstB + hStrideV210;
            auto srcU = srcSlice[1] + vIndex * hStrideYUV420P / 2;
            auto srcV = srcSlice[2] + vIndex * hStrideYUV420P / 2;

            for(int hIndex = 0; hIndex < hStrideYUV420P / 6; hIndex++){
                // Get lumas from above line
                auto y0 = *srcB++ << 2U;
                auto y1 = *srcB++ << 2U;
                auto y2 = *srcB++ << 2U;
                auto y3 = *srcB++ << 2U;
                auto y4 = *srcB++ << 2U;
                auto y5 = *srcB++ << 2U;

                // Get lumas from below line
                auto y0b = *srcBb++ << 2U;
                auto y1b = *srcBb++ << 2U;
                auto y2b = *srcBb++ << 2U;
                auto y3b = *srcBb++ << 2U;
                auto y4b = *srcBb++ << 2U;
                auto y5b = *srcBb++ << 2U;

                // Get chroma U
                auto u0 = *srcU++ << 2U;
                auto u1 = *srcU++ << 2U;
                auto u2 = *srcU++ << 2U;

                // Get chroma V
                auto v0 = *srcV++ << 2U;
                auto v1 = *srcV++ << 2U;
                auto v2 = *srcV++ << 2U;

                // Assign above line
                *dstB++ = (v0 << 20U) | (y0 << 10U) | u0;
                *dstB++ = (y2 << 20U) | (u1 << 10U) | y1;
                *dstB++ = (u2 << 20U) | (y3 << 10U) | v1;
                *dstB++ = (y5 << 20U) | (v2 << 10U) | y4;

                // Assign below line
                *dstBb++ = (v0 << 20U) | (y0b << 10U) | u0;
                *dstBb++ = (y2b << 20U) | (u1 << 10U) | y1b;
                *dstBb++ = (u2 << 20U) | (y3b << 10U) | v1;
                *dstBb++ = (y5b << 20U) | (v2 << 10U) | y4b;
            }
        }

        return;
    }
    #pragma endregion

    #pragma region NV12
    if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_UYVY422){
        // Used metrics
        int vStrideNV12 = height;
        int hStrideNV12 = width;
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;

        // Iterate blocks of 2x2 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideNV12 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * 2 * hStrideNV12;
            auto srcBb = srcB + hStrideNV12;
            auto dstB = dstSlice[0] + vIndex * 2 * hStrideUYVY422;
            auto dstBb = dstB + hStrideUYVY422;
            auto srcC = srcSlice[1] + vIndex * hStrideNV12;

            for(int hIndex = 0; hIndex < hStrideNV12 / 2; hIndex++){
                // Get chroma values
                uint8_t u = *srcC++; // U
                uint8_t v = *srcC++; // V

                                     // Assign above line values
                *dstB++ = u; // U0
                *dstB++ = *srcB++; // Y0
                *dstB++ = v; // V0
                *dstB++ = *srcB++; // Y1

                                   // Assign below line values
                *dstBb++ = u; // U0
                *dstBb++ = *srcBb++; // Y0
                *dstBb++ = v; // V0
                *dstBb++ = *srcBb++; // Y1
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_YUV422P){
        // Used metrics
        int vStrideNV12 = height;
        int hStrideNV12 = width;
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        int hStrideYUV422PChroma = hStrideYUV422P / 2;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideNV12 * hStrideNV12);

            // Iterate blocks of 2x2 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideNV12 / 2; vIndex++){
                // Discover buffer pointers
                auto srcC = srcSlice[1] + vIndex * hStrideNV12;
                auto dstU = dstSlice[1] + vIndex * 2 * hStrideYUV422PChroma;
                auto dstV = dstSlice[2] + vIndex * 2 * hStrideYUV422PChroma;
                auto dstUb = dstU + hStrideYUV422PChroma;
                auto dstVb = dstV + hStrideYUV422PChroma;

                for(int hIndex = 0; hIndex < hStrideNV12 / 2; hIndex++){
                    // Get chroma values
                    uint8_t u = *srcC++; // U
                    uint8_t v = *srcC++; // V

                                         // Assign values dupicated
                    *dstU++ = u;
                    *dstV++ = v;

                    *dstUb++ = u;
                    *dstVb++ = v;
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_YUV420P){
        // Used metrics
        int vStrideNV12 = height;
        int hStrideNV12 = width;
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;

        #pragma omp parallel
        {
            // Luma plane is the same
            #pragma omp single nowait
            memcpy(dstSlice[0], srcSlice[0], vStrideNV12 * hStrideNV12);

            // Iterate blocks of 2x2 channel points
            #pragma omp for schedule(static)
            for(int vIndex = 0; vIndex < vStrideNV12; vIndex++){
                // Discover buffer pointers
                auto srcC = srcSlice[1] + vIndex * hStrideNV12 / 2;
                auto dstU = dstSlice[1] + vIndex * hStrideYUV420P / 4;
                auto dstV = dstSlice[2] + vIndex * hStrideYUV420P / 4;

                for(int hIndex = 0; hIndex < hStrideNV12 / 4; hIndex++){
                    *dstU++ = *srcC++; // U
                    *dstV++ = *srcC++; // V
                }
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_NV12){
        // Used metrics
        int vStrideNV12 = height;
        int hStrideNV12 = width;

        // Copy data
        memcpy(dstSlice[0], srcSlice[0], vStrideNV12 * hStrideNV12);
        memcpy(dstSlice[1], srcSlice[1], vStrideNV12 * hStrideNV12 / 2);

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_NV12 && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideNV12 = height;
        int hStrideNV12 = width;
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Iterate blocks of 2x2 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideNV12 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * 2 * hStrideNV12;
            auto srcBb = srcB + hStrideNV12;
            auto dstB = reinterpret_cast<uint32_t*>(dstSlice[0]) + vIndex * 2 * hStrideV210;
            auto dstBb = dstB + hStrideV210;
            auto srcC = srcSlice[1] + vIndex * hStrideNV12;

            for(int hIndex = 0; hIndex < hStrideNV12 / 6; hIndex++){
                // Get lumas from above line
                auto y0 = *srcB++ << 2U;
                auto y1 = *srcB++ << 2U;
                auto y2 = *srcB++ << 2U;
                auto y3 = *srcB++ << 2U;
                auto y4 = *srcB++ << 2U;
                auto y5 = *srcB++ << 2U;

                // Get lumas from below line
                auto y0b = *srcBb++ << 2U;
                auto y1b = *srcBb++ << 2U;
                auto y2b = *srcBb++ << 2U;
                auto y3b = *srcBb++ << 2U;
                auto y4b = *srcBb++ << 2U;
                auto y5b = *srcBb++ << 2U;

                // Get chroma U and V
                auto u0 = *srcC++ << 2U;
                auto v0 = *srcC++ << 2U;
                auto u1 = *srcC++ << 2U;
                auto v1 = *srcC++ << 2U;
                auto u2 = *srcC++ << 2U;
                auto v2 = *srcC++ << 2U;

                // Assign above line
                *dstB++ = (v0 << 20U) | (y0 << 10U) | u0;
                *dstB++ = (y2 << 20U) | (u1 << 10U) | y1;
                *dstB++ = (u2 << 20U) | (y3 << 10U) | v1;
                *dstB++ = (y5 << 20U) | (v2 << 10U) | y4;

                // Assign below line
                *dstBb++ = (v0 << 20U) | (y0b << 10U) | u0;
                *dstBb++ = (y2b << 20U) | (u1 << 10U) | y1b;
                *dstBb++ = (u2 << 20U) | (y3b << 10U) | v1;
                *dstBb++ = (y5b << 20U) | (v2 << 10U) | y4b;
            }
        }

        return;
    }
    #pragma endregion

    #pragma region V210
    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_UYVY422){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;
        int vStrideUYVY422 = height;
        int hStrideUYVY422 = width * 2;

        // Iterate blocks of 1x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideV210; vIndex++){
            // Discover buffer pointers
            auto srcB = reinterpret_cast<uint32_t*>(srcSlice[0]) + vIndex * hStrideV210;
            auto dstB = dstSlice[0] + vIndex * hStrideUYVY422;

            for(int hIndex = 0; hIndex < hStrideV210 / 4; hIndex++){
                auto u0 = (*srcB >> 2U) & 0xFF; // U0
                auto y0 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y0
                auto v0 = ((*srcB >> 2U) >> 20U) & 0xFF; // V0
                *srcB++;

                auto y1 = (*srcB >> 2U) & 0xFF; // Y1
                auto u1 = ((*srcB >> 2U) >> 10U) & 0xFF; // U1
                auto y2 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y2
                *srcB++;

                auto v1 = (*srcB >> 2U) & 0xFF; // V1
                auto y3 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y3
                auto u2 = ((*srcB >> 2U) >> 20U) & 0xFF; // U2
                *srcB++;

                auto y4 = (*srcB >> 2U) & 0xFF; // Y4
                auto v2 = ((*srcB >> 2U) >> 10U) & 0xFF; // V2
                auto y5 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y5
                *srcB++;

                *(dstB++) = u0;
                *(dstB++) = y0;
                *(dstB++) = v0;
                *(dstB++) = y1;

                *(dstB++) = u1;
                *(dstB++) = y2;
                *(dstB++) = v1;
                *(dstB++) = y3;

                *(dstB++) = u2;
                *(dstB++) = y4;
                *(dstB++) = v2;
                *(dstB++) = y5;
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV422P){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        // Iterate blocks of 1x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideV210; vIndex++){
            // Discover buffer pointers
            auto srcB = reinterpret_cast<uint32_t*>(srcSlice[0]) + vIndex * hStrideV210;
            auto dstB = dstSlice[0] + vIndex * hStrideYUV422P;
            auto dstU = dstSlice[1] + vIndex * hStrideYUV422P / 2;
            auto dstV = dstSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideV210 / 4; hIndex++){
                auto u0 = (*srcB >> 2U) & 0xFF; // U0
                auto y0 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y0
                auto v0 = ((*srcB >> 2U) >> 20U) & 0xFF; // V0
                *srcB++;

                auto y1 = (*srcB >> 2U) & 0xFF; // Y1
                auto u1 = ((*srcB >> 2U) >> 10U) & 0xFF; // U1
                auto y2 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y2
                *srcB++;

                auto v1 = (*srcB >> 2U) & 0xFF; // V1
                auto y3 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y3
                auto u2 = ((*srcB >> 2U) >> 20U) & 0xFF; // U2
                *srcB++;

                auto y4 = (*srcB >> 2U) & 0xFF; // Y4
                auto v2 = ((*srcB >> 2U) >> 10U) & 0xFF; // V2
                auto y5 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y5
                *srcB++;

                *(dstU++) = u0;
                *(dstB++) = y0;
                *(dstV++) = v0;
                *(dstB++) = y1;

                *(dstU++) = u1;
                *(dstB++) = y2;
                *(dstV++) = v1;
                *(dstB++) = y3;

                *(dstU++) = u2;
                *(dstB++) = y4;
                *(dstV++) = v2;
                *(dstB++) = y5;
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV420P){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;
        int vStrideYUV420P = height;
        int hStrideYUV420P = width;

        // Iterate blocks of 2x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideV210 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = reinterpret_cast<uint32_t*>(srcSlice[0]) + vIndex * 2 * hStrideV210;
            auto srcBb = srcB + hStrideV210;
            auto dstB = dstSlice[0] + vIndex * 2 * hStrideYUV420P;
            auto dstBb = dstB + hStrideYUV420P;
            auto dstU = dstSlice[1] + vIndex * hStrideYUV420P / 2;
            auto dstV = dstSlice[2] + vIndex * hStrideYUV420P / 2;

            for(int hIndex = 0; hIndex < hStrideV210 / 4; hIndex++){
                // Get above line
                auto u0 = (*srcB >> 2U) & 0xFF; // U0
                auto y0 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y0
                auto v0 = ((*srcB >> 2U) >> 20U) & 0xFF; // V0
                *srcB++;

                auto y1 = (*srcB >> 2U) & 0xFF; // Y1
                auto u1 = ((*srcB >> 2U) >> 10U) & 0xFF; // U1
                auto y2 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y2
                *srcB++;

                auto v1 = (*srcB >> 2U) & 0xFF; // V1
                auto y3 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y3
                auto u2 = ((*srcB >> 2U) >> 20U) & 0xFF; // U2
                *srcB++;

                auto y4 = (*srcB >> 2U) & 0xFF; // Y4
                auto v2 = ((*srcB >> 2U) >> 10U) & 0xFF; // V2
                auto y5 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y5
                *srcB++;

                // Get below line
                auto u0b = (*srcBb >> 2U) & 0xFF; // U0
                auto y0b = ((*srcBb >> 2U) >> 10U) & 0xFF; // Y0
                auto v0b = ((*srcBb >> 2U) >> 20U) & 0xFF; // V0
                *srcBb++;

                auto y1b = (*srcBb >> 2U) & 0xFF; // Y1
                auto u1b = ((*srcBb >> 2U) >> 10U) & 0xFF; // U1
                auto y2b = ((*srcBb >> 2U) >> 20U) & 0xFF; // Y2
                *srcBb++;

                auto v1b = (*srcBb >> 2U) & 0xFF; // V1
                auto y3b = ((*srcBb >> 2U) >> 10U) & 0xFF; // Y3
                auto u2b = ((*srcBb >> 2U) >> 20U) & 0xFF; // U2
                *srcBb++;

                auto y4b = (*srcBb >> 2U) & 0xFF; // Y4
                auto v2b = ((*srcBb >> 2U) >> 10U) & 0xFF; // V2
                auto y5b = ((*srcBb >> 2U) >> 20U) & 0xFF; // Y5
                *srcBb++;

                // Assign above luma values
                *dstB++ = y0;
                *dstB++ = y1;
                *dstB++ = y2;
                *dstB++ = y3;
                *dstB++ = y4;
                *dstB++ = y5;

                // Assign below luma values
                *dstBb++ = y0b;
                *dstBb++ = y1b;
                *dstBb++ = y2b;
                *dstBb++ = y3b;
                *dstBb++ = y4b;
                *dstBb++ = y5b;

                // Assign chroma values
                *dstU++ = uint8_t(roundFast((static_cast<double>(u0) + static_cast<double>(u0b)) / 2.));
                *dstU++ = uint8_t(roundFast((static_cast<double>(u1) + static_cast<double>(u1b)) / 2.));
                *dstU++ = uint8_t(roundFast((static_cast<double>(u2) + static_cast<double>(u2b)) / 2.));

                *dstV++ = uint8_t(roundFast((static_cast<double>(v0) + static_cast<double>(v0b)) / 2.));
                *dstV++ = uint8_t(roundFast((static_cast<double>(v1) + static_cast<double>(v1b)) / 2.));
                *dstV++ = uint8_t(roundFast((static_cast<double>(v2) + static_cast<double>(v2b)) / 2.));
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_NV12){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;
        int vStrideNV12 = height;
        int hStrideNV12 = width;

        // Iterate blocks of 2x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideV210 / 2; vIndex++){
            // Discover buffer pointers
            auto srcB = reinterpret_cast<uint32_t*>(srcSlice[0]) + vIndex * 2 * hStrideV210;
            auto srcBb = srcB + hStrideV210;
            auto dstB = dstSlice[0] + vIndex * 2 * hStrideNV12;
            auto dstBb = dstB + hStrideNV12;
            auto dstC = dstSlice[1] + vIndex * hStrideNV12;

            for(int hIndex = 0; hIndex < hStrideV210 / 4; hIndex++){
                // Get above line
                auto u0 = (*srcB >> 2U) & 0xFF; // U0
                auto y0 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y0
                auto v0 = ((*srcB >> 2U) >> 20U) & 0xFF; // V0
                *srcB++;

                auto y1 = (*srcB >> 2U) & 0xFF; // Y1
                auto u1 = ((*srcB >> 2U) >> 10U) & 0xFF; // U1
                auto y2 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y2
                *srcB++;

                auto v1 = (*srcB >> 2U) & 0xFF; // V1
                auto y3 = ((*srcB >> 2U) >> 10U) & 0xFF; // Y3
                auto u2 = ((*srcB >> 2U) >> 20U) & 0xFF; // U2
                *srcB++;

                auto y4 = (*srcB >> 2U) & 0xFF; // Y4
                auto v2 = ((*srcB >> 2U) >> 10U) & 0xFF; // V2
                auto y5 = ((*srcB >> 2U) >> 20U) & 0xFF; // Y5
                *srcB++;

                // Get below line
                auto u0b = (*srcBb >> 2U) & 0xFF; // U0
                auto y0b = ((*srcBb >> 2U) >> 10U) & 0xFF; // Y0
                auto v0b = ((*srcBb >> 2U) >> 20U) & 0xFF; // V0
                *srcBb++;

                auto y1b = (*srcBb >> 2U) & 0xFF; // Y1
                auto u1b = ((*srcBb >> 2U) >> 10U) & 0xFF; // U1
                auto y2b = ((*srcBb >> 2U) >> 20U) & 0xFF; // Y2
                *srcBb++;

                auto v1b = (*srcBb >> 2U) & 0xFF; // V1
                auto y3b = ((*srcBb >> 2U) >> 10U) & 0xFF; // Y3
                auto u2b = ((*srcBb >> 2U) >> 20U) & 0xFF; // U2
                *srcBb++;

                auto y4b = (*srcBb >> 2U) & 0xFF; // Y4
                auto v2b = ((*srcBb >> 2U) >> 10U) & 0xFF; // V2
                auto y5b = ((*srcBb >> 2U) >> 20U) & 0xFF; // Y5
                *srcBb++;

                // Assign above luma values
                *dstB++ = y0;
                *dstB++ = y1;
                *dstB++ = y2;
                *dstB++ = y3;
                *dstB++ = y4;
                *dstB++ = y5;

                // Assign below luma values
                *dstBb++ = y0b;
                *dstBb++ = y1b;
                *dstBb++ = y2b;
                *dstBb++ = y3b;
                *dstBb++ = y4b;
                *dstBb++ = y5b;

                // Assign chroma values
                *dstC++ = uint8_t(roundFast((static_cast<double>(u0) + static_cast<double>(u0b)) / 2.));
                *dstC++ = uint8_t(roundFast((static_cast<double>(v0) + static_cast<double>(v0b)) / 2.));
                *dstC++ = uint8_t(roundFast((static_cast<double>(u1) + static_cast<double>(u1b)) / 2.));
                *dstC++ = uint8_t(roundFast((static_cast<double>(v1) + static_cast<double>(v1b)) / 2.));
                *dstC++ = uint8_t(roundFast((static_cast<double>(u2) + static_cast<double>(u2b)) / 2.));
                *dstC++ = uint8_t(roundFast((static_cast<double>(v2) + static_cast<double>(v2b)) / 2.));
            }
        }

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Copy data
        memcpy(dstSlice[0], srcSlice[0], vStrideV210 * hStrideV210 * sizeof(uint32_t));

        return;
    }

    if(srcPixelFormat == AV_PIX_FMT_V210 && dstPixelFormat == AV_PIX_FMT_YUV422PNORM){
        // Used metrics
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;

        // Create const for normalization
        double constLuma = 219. / 1023.;
        double constChroma = 224. / 1023.;
        double const16 = 16.;

        // Iterate blocks of 1x4 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideV210; vIndex++){
            // Discover buffer pointers
            auto srcB = reinterpret_cast<uint32_t*>(srcSlice[0]) + vIndex * hStrideV210;
            auto dstB = dstSlice[0] + vIndex * hStrideYUV422P;
            auto dstU = dstSlice[1] + vIndex * hStrideYUV422P / 2;
            auto dstV = dstSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideV210 / 4; hIndex++){
                auto u0 = *srcB & 0x3FF; // U0
                auto y0 = (*srcB >> 10U) & 0x3FF; // Y0
                auto v0 = (*srcB >> 20U) & 0x3FF; // V0
                *srcB++;

                auto y1 = *srcB & 0x3FF; // Y1
                auto u1 = (*srcB >> 10U) & 0x3FF; // U1
                auto y2 = (*srcB >> 20U) & 0x3FF; // Y2
                *srcB++;

                auto v1 = *srcB & 0x3FF; // V1
                auto y3 = (*srcB >> 10U) & 0x3FF; // Y3
                auto u2 = (*srcB >> 20U) & 0x3FF; // U2
                *srcB++;

                auto y4 = *srcB & 0x3FF; // Y4
                auto v2 = (*srcB >> 10U) & 0x3FF; // V2
                auto y5 = (*srcB >> 20U) & 0x3FF; // Y5
                *srcB++;

                *dstU++ = uint8_t(roundFast(static_cast<double>(u0) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y0) * constLuma + const16));
                *dstV++ = uint8_t(roundFast(static_cast<double>(v0) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y1) * constLuma + const16));

                *dstU++ = uint8_t(roundFast(static_cast<double>(u1) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y2) * constLuma + const16));
                *dstV++ = uint8_t(roundFast(static_cast<double>(v1) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y3) * constLuma + const16));

                *dstU++ = uint8_t(roundFast(static_cast<double>(u2) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y4) * constLuma + const16));
                *dstV++ = uint8_t(roundFast(static_cast<double>(v2) * constChroma + const16));
                *dstB++ = uint8_t(roundFast(static_cast<double>(y5) * constLuma + const16));
            }
        }

        return;
    }
    #pragma endregion

    #pragma region YUV422PNORM
    if(srcPixelFormat == AV_PIX_FMT_YUV422PNORM && dstPixelFormat == AV_PIX_FMT_V210){
        // Used metrics
        int vStrideYUV422P = height;
        int hStrideYUV422P = width;
        int vStrideV210 = height;
        int hStrideV210 = width / 6 * 4;

        // Create const for normalization
        double const16 = 16.;
        double constLuma = 1023. / 219.;
        double constChroma = 1023. / 224.;

        // Iterate blocks of 1x6 channel points
        #pragma omp parallel for schedule(static)
        for(int vIndex = 0; vIndex < vStrideYUV422P; vIndex++){
            // Discover buffer pointers
            auto srcB = srcSlice[0] + vIndex * hStrideYUV422P;
            auto dstB = reinterpret_cast<uint32_t*>(dstSlice[0]) + vIndex * hStrideV210;
            auto srcU = srcSlice[1] + vIndex * hStrideYUV422P / 2;
            auto srcV = srcSlice[2] + vIndex * hStrideYUV422P / 2;

            for(int hIndex = 0; hIndex < hStrideYUV422P / 6; hIndex++){
                // Get components from source
                auto u0n = *srcU++; // U0
                auto y0n = *srcB++; // Y0
                auto v0n = *srcV++; // V0
                auto y1n = *srcB++; // Y1

                auto u1n = *srcU++; // U1
                auto y2n = *srcB++; // Y2
                auto v1n = *srcV++; // V1
                auto y3n = *srcB++; // Y3

                auto u2n = *srcU++; // U2
                auto y4n = *srcB++; // Y4
                auto v2n = *srcV++; // V2
                auto y5n = *srcB++; // Y5

                                    // Denormalize values
                auto v0 = uint16_t(roundFast((static_cast<double>(v0n) - const16) * constChroma)) & 0x3FF;
                auto y0 = uint16_t(roundFast((static_cast<double>(y0n) - const16) * constLuma)) & 0x3FF;
                auto u0 = uint16_t(roundFast((static_cast<double>(u0n) - const16) * constChroma)) & 0x3FF;
                auto y2 = uint16_t(roundFast((static_cast<double>(y2n) - const16) * constLuma)) & 0x3FF;

                auto u1 = uint16_t(roundFast((static_cast<double>(u1n) - const16) * constChroma)) & 0x3FF;
                auto y1 = uint16_t(roundFast((static_cast<double>(y1n) - const16) * constLuma)) & 0x3FF;
                auto u2 = uint16_t(roundFast((static_cast<double>(u2n) - const16) * constChroma)) & 0x3FF;
                auto y3 = uint16_t(roundFast((static_cast<double>(y3n) - const16) * constLuma)) & 0x3FF;

                auto v1 = uint16_t(roundFast((static_cast<double>(v1n) - const16) * constChroma)) & 0x3FF;
                auto y5 = uint16_t(roundFast((static_cast<double>(y5n) - const16) * constLuma)) & 0x3FF;
                auto v2 = uint16_t(roundFast((static_cast<double>(v2n) - const16) * constChroma)) & 0x3FF;
                auto y4 = uint16_t(roundFast((static_cast<double>(y4n) - const16) * constLuma)) & 0x3FF;

                // Assign value
                *dstB++ = (v0 << 20U) | (y0 << 10U) | u0;
                *dstB++ = (y2 << 20U) | (u1 << 10U) | y1;
                *dstB++ = (u2 << 20U) | (y3 << 10U) | v1;
                *dstB++ = (y5 << 20U) | (v2 << 10U) | y4;
            }
        }

        return;
    }
    #pragma endregion
}

// Precalculate coefficients
int cuda_omp_preCalculateCoefficients(int srcSize, int dstSize, int operation, int pixelSupport, double(*coefFunc)(double), float* &preCalculatedCoefs){
    // Calculate size ratio
    double sizeRatio = static_cast<double>(dstSize) / static_cast<double>(srcSize);

    // Calculate once
    double pixelSupportDiv2 = pixelSupport / 2.;
    bool isDownScale = sizeRatio < 1.;
    double regionRadius = isDownScale ? pixelSupportDiv2 / sizeRatio : pixelSupportDiv2;
    double filterStep = isDownScale && operation != SWS_POINT ? 1. / sizeRatio : 1.;
    int numCoefficients = isDownScale ? ceil(pixelSupport / sizeRatio) : pixelSupport;
    int numCoefficientsDiv2 = numCoefficients / 2;

    // Calculate number of lines of coefficients
    int preCalcCoefSize = isDownScale ? (lcm(srcSize, dstSize) / min(srcSize, dstSize)) * (static_cast<double>(srcSize) / static_cast<double>(dstSize)) : lcm(srcSize, dstSize) / min(srcSize, dstSize);

    // Initialize array
    preCalculatedCoefs = static_cast<float*>(malloc(preCalcCoefSize * numCoefficients * sizeof(float)));

    // For each necessary line of coefficients
    for(int col = 0; col < preCalcCoefSize; col++){
        // Calculate once
        int indexOffset = col * numCoefficients;

        // Original line index coordinate
        double colOriginal = (static_cast<double>(col) + .5) / sizeRatio;

        // Discover source limit pixels
        double nearPixel = colOriginal - filterStep;
        double leftPixel = colOriginal - regionRadius;

        // Discover offset to pixel of filter start
        double offset = round(leftPixel) + .5 - leftPixel;
        // Calculate maximum distance to normalize distances
        double maxDistance = colOriginal - nearPixel;
        // Calculate where filtering will start
        double startPosition = leftPixel + offset;

        // Calculate coefficients
        float coefAcc = 0.f;
        for(int index = 0; index < numCoefficients; index++){
            float coefHolder = static_cast<float>(coefFunc((colOriginal - (startPosition + index)) / maxDistance));
            coefAcc += coefHolder;
            preCalculatedCoefs[indexOffset + index] = coefHolder;
        }

        // Avoid lines of coefficients without valid values
        if(operation == SWS_POINT){
            if(preCalculatedCoefs[indexOffset + numCoefficientsDiv2 - 1] == preCalculatedCoefs[indexOffset + numCoefficientsDiv2]){
                if(isDownScale){
                    if(preCalculatedCoefs[indexOffset + numCoefficientsDiv2 - 1] == 0.f && numCoefficients % 2 != 0)
                        preCalculatedCoefs[indexOffset + numCoefficientsDiv2 - 1] = 1.f;
                    else
                        preCalculatedCoefs[indexOffset + numCoefficientsDiv2] = 1.f;
                } else
                    preCalculatedCoefs[indexOffset + numCoefficientsDiv2] = 1.f;
            }
        }

        // Normalizes coefficients except on Nearest Neighbor interpolation
        if(operation != SWS_POINT)
            for(int index = 0; index < numCoefficients; index++)
                preCalculatedCoefs[indexOffset + index] /= coefAcc;
    }

    // Success
    return preCalcCoefSize;
}

// Change the image dimension
__global__ void cuda_resize(int srcWidth, int srcHeight, int dstWidth, int dstHeight,
    float scaleWidthRatio, float scaleHeightRatio, uint8_t* srcData, uint8_t* dstData, float regionHRadius, float regionVRadius, int colorChannel,
    int vCoefsSize, int numVCoefs, float* vCoefs, int hCoefsSize, int numHCoefs, float* hCoefs){

    // Calculate pixel location
    int lin = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate once coefficients index
    int linCoefOffset = (lin % vCoefsSize) * numVCoefs;
    int colCoefOffset = (col % hCoefsSize) * numHCoefs;

    // Original index coordinates
    float linOriginal = (static_cast<float>(lin) + .5f) / scaleHeightRatio;
    float colOriginal = (static_cast<float>(col) + .5f) / scaleWidthRatio;

    // Discover source limit pixels
    float upperPixel = linOriginal - regionVRadius;
    float leftPixel = colOriginal - regionHRadius;

    // Discover offset to pixel of filter start
    float offsetV = roundf(upperPixel) + .5f - upperPixel;
    float offsetH = roundf(leftPixel) + .5f - leftPixel;

    // Calculate once
    float startLinPosition = upperPixel + offsetV;
    float startColPosition = leftPixel + offsetH;

    // Color accumulator
    float acc = 0.f;
    // Calculate resulting color from coefficients
    for(int indexV = 0; indexV < numVCoefs; indexV++){
        // Get vertical coefficient
        float vCoef = vCoefs[linCoefOffset + indexV];

        // Calculate source pixel line index
        int srcLinIndex = startLinPosition + indexV;
        // Clamp coords
        if(srcLinIndex < 0)
            srcLinIndex = 0;
        else if(srcLinIndex > srcHeight - 1)
            srcLinIndex = srcHeight - 1;

        // Calculate once
        int srcLinIndexOffset = srcLinIndex * srcWidth;

        for(int indexH = 0; indexH < numHCoefs; indexH++){
            // Get horizontal coefficient
            float hCoef = hCoefs[colCoefOffset + indexH];

            // Calculate source pixel column index
            int srcColIndex = startColPosition + indexH;
            // Clamp coords
            if(srcColIndex < 0)
                srcColIndex = 0;
            else if(srcColIndex > srcWidth - 1)
                srcColIndex = srcWidth - 1;

            // Get neighbor pixel color
            uint8_t colorHolder = srcData[srcLinIndexOffset + srcColIndex];

            // Calculate pixel color weight
            float weight = vCoef * hCoef;

            // Weighted color
            acc += colorHolder * weight;
        }
    }

    // Clamp value to avoid undershooting and overshooting
    if(colorChannel == 0){
        if(acc < 16.f)
            acc = 16.f;
        else if(acc > 235.f)
            acc = 235.f;
    } else{
        if(acc < 16.f)
            acc = 16.f;
        else if(acc > 240.f)
            acc = 240.f;
    }

    // Assign calculated color to destiantion data
    dstData[lin * dstWidth + col] = uint8_t(lroundf(acc));
}

// Prepares the resample operation
void cuda_resample_aux(AVFrame* src, AVFrame* dst, int operation){
    // Access once
    int srcWidth = src->width, srcHeight = src->height;
    int srcFormat = src->format;
    int dstWidth = dst->width, dstHeight = dst->height;
    int dstFormat = dst->format;

    // Check if is only a format conversion
    bool isOnlyFormatConversion = srcWidth == dstWidth && srcHeight == dstHeight;
    // Changes image pixel format only
    if(isOnlyFormatConversion && false){
        // Format conversion operation
        cuda_omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, dstFormat, dst->data);
        // End resample operation
        return;
    }

    // Get standard supported pixel format in scaling
    int scaleFormat = getScaleFormat(srcFormat, dstFormat);

    // Get scale ratios
    float scaleHeightRatio = static_cast<float>(dstHeight) / static_cast<float>(srcHeight);
    float scaleWidthRatio = static_cast<float>(dstWidth) / static_cast<float>(srcWidth);

    // Needed resources for coefficients calculations
    double(*coefFunc)(double) = getCoefMethod(operation);
    int pixelSupport = getPixelSupport(operation);

    // Calculate once
    float pixelSupportDiv2 = pixelSupport / 2.f;
    bool isDownScaleV = scaleHeightRatio < 1.f;
    bool isDownScaleH = scaleWidthRatio < 1.f;
    float regionVRadius = isDownScaleV ? pixelSupportDiv2 / scaleHeightRatio : pixelSupportDiv2;
    float regionHRadius = isDownScaleH ? pixelSupportDiv2 / scaleWidthRatio : pixelSupportDiv2;
    int numVCoefs = isDownScaleV ? ceil(pixelSupport / scaleHeightRatio) : pixelSupport;
    int numHCoefs = isDownScaleH ? ceil(pixelSupport / scaleWidthRatio) : pixelSupport;

    // Chroma size discovery
    float widthPerc = 1.f;
    float heightPerc = 1.f;
    if(scaleFormat == AV_PIX_FMT_YUV422P || scaleFormat == AV_PIX_FMT_YUV420P || scaleFormat == AV_PIX_FMT_YUV422PNORM)
        widthPerc = .5f;
    if(scaleFormat == AV_PIX_FMT_YUV420P)
        heightPerc = .5f;

    // Precalculate coefficients
    float* vCoefsHost;
    int vCoefsSize = cuda_omp_preCalculateCoefficients(srcHeight, dstHeight, operation, pixelSupport, coefFunc, vCoefsHost);
    float* hCoefsHost;
    int hCoefsSize = cuda_omp_preCalculateCoefficients(srcWidth, dstWidth, operation, pixelSupport, coefFunc, hCoefsHost);

    // Allocate coefficients buffer in device
    float *vCoefsDevice, *hCoefsDevice;
    hipMalloc((void **) &vCoefsDevice, vCoefsSize * numVCoefs * sizeof(float));
    hipMalloc((void **) &hCoefsDevice, hCoefsSize * numHCoefs * sizeof(float));

    // Copy coefficients to device
    hipMemcpy(vCoefsDevice, vCoefsHost, vCoefsSize * numVCoefs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(hCoefsDevice, hCoefsHost, hCoefsSize * numHCoefs * sizeof(float), hipMemcpyHostToDevice);

    // Free host coefficients
    free(vCoefsHost);
    free(hCoefsHost);

    // Temporary buffer
    uint8_t** forScalePointersHost;
    // Allocate channel buffer pointers
    allocBuffers(forScalePointersHost, srcWidth, srcHeight, scaleFormat);

    // Resamples image to a supported format
    cuda_omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, scaleFormat, forScalePointersHost);

    // Create target buffer in device
    uint8_t** forScalePointersDevice;
    int* forScalePointersDeviceSizes;
    // Allocate source buffer in device
    cudaAllocBuffers(forScalePointersDevice, forScalePointersDeviceSizes, srcWidth, srcHeight, scaleFormat);

    // Copy source data to device
    cudaCopyBuffersToGPU(forScalePointersHost, forScalePointersDevice, forScalePointersDeviceSizes);

    // Free host memory
    free2dBuffer(forScalePointersHost, 3);

    // Create target buffer in device
    uint8_t** fromScalePointersDevice;
    int* fromScalePointersDeviceSizes;
    // Allocate source buffer in device
    cudaAllocBuffers(fromScalePointersDevice, fromScalePointersDeviceSizes, dstWidth, dstHeight, scaleFormat);

    // Create launch parameters of resize kernel
    pair<dim3, dim3> resizeLP = calculateResizeLP(dstWidth, dstHeight, 32);
    // Recalculate launch parameters for chromas
    int heightChroma = static_cast<int>(dstHeight * heightPerc);
    int widthChroma = static_cast<int>(dstWidth * widthPerc);
    pair<dim3, dim3> resizeChromaLP = calculateResizeLP(widthChroma, heightChroma, 32);

    // Apply the resizing operation to luma channel
    cuda_resize << <resizeLP.first, resizeLP.second>> > (srcWidth, srcHeight, dstWidth, dstHeight,
        scaleWidthRatio, scaleHeightRatio, forScalePointersDevice[0], fromScalePointersDevice[0], regionHRadius, regionVRadius, 0,
        vCoefsSize, numVCoefs, vCoefsDevice, hCoefsSize, numHCoefs, hCoefsDevice);

    // Apply the resizing operation to U chroma channel
    cuda_resize << <resizeChromaLP.first, resizeChromaLP.second>> > (static_cast<int>(srcWidth * widthPerc), static_cast<int>(srcHeight * heightPerc), widthChroma, heightChroma,
        scaleWidthRatio, scaleHeightRatio, forScalePointersDevice[1], fromScalePointersDevice[1], regionHRadius, regionVRadius, 1,
        vCoefsSize, numVCoefs, vCoefsDevice, hCoefsSize, numHCoefs, hCoefsDevice);

    // Apply the resizing operation to V chroma channel
    cuda_resize << <resizeChromaLP.first, resizeChromaLP.second >> > (static_cast<int>(srcWidth * widthPerc), static_cast<int>(srcHeight * heightPerc), widthChroma, heightChroma,
        scaleWidthRatio, scaleHeightRatio, forScalePointersDevice[2], fromScalePointersDevice[2], regionHRadius, regionVRadius, 2,
        vCoefsSize, numVCoefs, vCoefsDevice, hCoefsSize, numHCoefs, hCoefsDevice);

    // Synchronize GPU
    hipDeviceSynchronize();

    // Free used data resources
    freeCudaMemory(forScalePointersDevice);
    free(forScalePointersDeviceSizes);

    // Temporary buffer
    uint8_t** fromScalePointersHost;
    // Allocate channel buffer pointers
    allocBuffers(fromScalePointersHost, dstWidth, dstHeight, scaleFormat);

    // Copy resulting data from device
    cudaCopyBuffersFromGPU(fromScalePointersHost, fromScalePointersDevice, fromScalePointersDeviceSizes);

    // Free used data resources
    freeCudaMemory(fromScalePointersDevice);
    free(fromScalePointersDeviceSizes);

    // Free coefficients in device
    hipFree(vCoefsDevice);
    hipFree(hCoefsDevice);

    // Resamples image to target format
    cuda_omp_formatConversion(dstWidth, dstHeight, scaleFormat, fromScalePointersHost, dstFormat, dst->data);

    // Free used resources
    free2dBuffer(fromScalePointersHost, 3);

    // Sucess
    return;
}

// Wrapper for the cuda resample operation method
int cuda_resample(AVFrame* src, AVFrame* dst, int operation){
    // Access once
    AVPixelFormat srcFormat = static_cast<AVPixelFormat>(src->format);
    AVPixelFormat dstFormat = static_cast<AVPixelFormat>(dst->format);

    // Verify valid frames
    if(src == nullptr || dst == nullptr){
        cerr << "[CUDA] One or both input frames are null!" << endl;
        return -1;
    }

    // Verify valid input data
    if(!src->data || !src->linesize || !dst->data || !dst->linesize){
        cerr << "[CUDA] Frame data buffers can not be null!" << endl;
        return -1;
    }

    // Verify valid input dimensions
    if(src->width < 0 || src->height < 0 || dst->width < 0 || dst->height < 0){
        cerr << "[CUDA] Frame dimensions can not be a negative number!" << endl;
        return -1;
    }

    // Verify if data is aligned
    if(((src->width % 4 != 0 && srcFormat == AV_PIX_FMT_UYVY422) || (dst->width % 4 != 0 && dstFormat == AV_PIX_FMT_UYVY422)) &&
        ((src->width % 12 != 0 && srcFormat == AV_PIX_FMT_V210) || (dst->width % 12 != 0 && dstFormat == AV_PIX_FMT_V210))){
        cerr << "[CUDA] Can not handle unaligned data!" << endl;
        return -1;
    }

    // Verify valid resize
    if((src->width < dst->width && src->height > dst->height) ||
        (src->width > dst->width && src->height < dst->height)){
        cerr << "[CUDA] Can not upscale in an orientation and downscale another!" << endl;
        return -1;
    }

    // Verify if supported conversion
    if(!hasSupportedConversion(srcFormat, dstFormat)){
        cerr << "[CUDA] Pixel format conversion is not supported!" << endl;
        return -1;
    }

    // Verify if supported scaling operation
    if(!isSupportedOperation(operation)){
        cerr << "[CUDA] Scaling operation is not supported" << endl;
        return -1;
    }

    // Variables used
    int duration = -1;
    high_resolution_clock::time_point initTime, stopTime;

    // Start counting operation execution time
    initTime = high_resolution_clock::now();

    // Apply the scaling operation
    cuda_resample_aux(src, dst, operation);

    // Stop counting operation execution time
    stopTime = high_resolution_clock::now();

    // Calculate the execution time
    duration = duration_cast<microseconds>(stopTime - initTime).count();

    // Return execution time of the scaling operation
    return duration;
}