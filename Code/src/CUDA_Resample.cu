#include "hip/hip_runtime.h"
#include "CUDA_Resample.h"

texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texY;
texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texU;
texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texV;

// Allocate image channels data buffers depending of the pixel format
void cudaAllocBuffers(uint8_t** &buffer, int* &bufferSize, int width, int height, int pixelFormat){
    // Allocate channel buffer size
    bufferSize = static_cast<int*>(malloc(3 * sizeof(int)));

    // Calculate once
    int wxh = width * height;
    int wxhDiv2 = wxh / 2;
    int wxhDiv4 = wxh / 4;

    // Calculate buffer sizes for each pixel format
    switch(pixelFormat){
        case AV_PIX_FMT_UYVY422:
            bufferSize[0] = wxh * 2;
            bufferSize[1] = 0;
            bufferSize[2] = 0;
            break;
        case AV_PIX_FMT_YUV422P:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv2;
            bufferSize[2] = wxhDiv2;
            break;
        case AV_PIX_FMT_YUV420P:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv4;
            bufferSize[2] = wxhDiv4;
            break;
        case AV_PIX_FMT_NV12:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv2;
            bufferSize[2] = 0;
            break;
        case AV_PIX_FMT_V210:
            bufferSize[0] = height * 128 * ((width + 47) / 48);
            bufferSize[1] = 0;
            bufferSize[2] = 0;
            break;
        case AV_PIX_FMT_YUV422PNORM:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv2;
            bufferSize[2] = wxhDiv2;
            break;
    }

    // Allocate buffer memory
    buffer = static_cast<uint8_t**>(malloc(3 * sizeof(uint8_t*)));

    // Allocate buffer in the GPU memory
    hipMalloc((void **) &buffer[0], bufferSize[0]);
    if(bufferSize[1] != 0)
        hipMalloc((void **) &buffer[1], bufferSize[1]);
    if(bufferSize[2] != 0)
        hipMalloc((void **) &buffer[2], bufferSize[2]);
}

// Free used GPU memory
void freeCudaMemory(uint8_t** &buffer){
    // Iterate each channel and free memory
    for(int i = 0; i < 3; i++)
        hipFree(buffer[i]);

    // Free host memory
    free(buffer);
}

// Copy data from host to device
void cudaCopyBuffersToGPU(uint8_t* srcBuffer[], uint8_t* gpuBuffer[], int* &bufferSize){
    // First channel
    hipMemcpy(gpuBuffer[0], srcBuffer[0], bufferSize[0], hipMemcpyHostToDevice);

    // Copy chroma channels if they exist
    if(bufferSize[1] != 0)
        hipMemcpy(gpuBuffer[1], srcBuffer[1], bufferSize[1], hipMemcpyHostToDevice);
    if(bufferSize[2] != 0)
        hipMemcpy(gpuBuffer[2], srcBuffer[2], bufferSize[2], hipMemcpyHostToDevice);
}

// Copy data from device to host
void cudaCopyBuffersFromGPU(uint8_t* targetBuffer[], uint8_t* gpuBuffer[], int* &bufferSize){
    // First channel
    hipMemcpy(targetBuffer[0], gpuBuffer[0], bufferSize[0], hipMemcpyDeviceToHost);

    // Copy chroma channels if they exist
    if(bufferSize[1] != 0)
        hipMemcpy(targetBuffer[1], gpuBuffer[1], bufferSize[1], hipMemcpyDeviceToHost);
    if(bufferSize[2] != 0)
        hipMemcpy(targetBuffer[2], gpuBuffer[2], bufferSize[2], hipMemcpyDeviceToHost);
}

// Calculate launch parameters of resize kernel
pair<dim3, dim3> calculateResizeLP(int width, int height){
    // Define the maximum number of thread dim1 size
    int maxNumThreads = 32;

    // Find the thead size
    int vThreadSize = min(maxNumThreads, greatestDivisor(height));
    int hThreadSize = min(maxNumThreads, greatestDivisor(width));

    // Calculate the block size
    int vBlockSize = height / vThreadSize;
    int hBlockSize = width / hThreadSize;

    // Return valid launch parameters
    return pair<dim3, dim3>(dim3(hBlockSize, vBlockSize), dim3(hThreadSize, vThreadSize));
}

// ------------------------------------------------------------------

// Nearest neighbor and bilinear hardware interpolation for tex y
__global__ void scaleTexY(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(tex2D(texY, colOriginal, linOriginal) * 255.f));
}

// Nearest neighbor and bilinear hardware interpolation for tex u
__global__ void scaleTexU(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(tex2D(texU, colOriginal, linOriginal) * 255.f));
}

// Nearest neighbor and bilinear hardware interpolation for tex v
__global__ void scaleTexV(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(tex2D(texV, colOriginal, linOriginal) * 255.f));
}

// Calculate coefficient of cubic interpolation
inline __device__ float cubicFilter(const float x, const float c0, const float c1, const float c2, const float c3){
    // Resulting color is the sum of all weighted colors
    float result = c0 * (-.6f * x * (x * (x - 2.f) + 1.f));
    result += c1 * (x * x * (1.4f * x - 2.4f) + 1.f);
    result += c2 * (x * (x * (-1.4f * x + 1.8f) + .6f));
    result += c3 * (.6f * x * x * (x - 1.f));
    return result;
}

// Bicubic interpolation for tex y
__global__ void cubicScaleY(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin - 1), tex2D(texY, pixCol, pixLin - 1), tex2D(texY, pixCol + 1, pixLin - 1), tex2D(texY, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin), tex2D(texY, pixCol, pixLin), tex2D(texY, pixCol + 1, pixLin), tex2D(texY, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin + 1), tex2D(texY, pixCol, pixLin + 1), tex2D(texY, pixCol + 1, pixLin + 1), tex2D(texY, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin + 2), tex2D(texY, pixCol, pixLin + 2), tex2D(texY, pixCol + 1, pixLin + 2), tex2D(texY, pixCol + 2, pixLin + 2)))));
}

// Bicubic interpolation for tex u
__global__ void cubicScaleU(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin - 1), tex2D(texU, pixCol, pixLin - 1), tex2D(texU, pixCol + 1, pixLin - 1), tex2D(texU, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin), tex2D(texU, pixCol, pixLin), tex2D(texU, pixCol + 1, pixLin), tex2D(texU, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin + 1), tex2D(texU, pixCol, pixLin + 1), tex2D(texU, pixCol + 1, pixLin + 1), tex2D(texU, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin + 2), tex2D(texU, pixCol, pixLin + 2), tex2D(texU, pixCol + 1, pixLin + 2), tex2D(texU, pixCol + 2, pixLin + 2)))));
}

// Bicubic interpolation for tex v
__global__ void cubicScaleV(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin - 1), tex2D(texV, pixCol, pixLin - 1), tex2D(texV, pixCol + 1, pixLin - 1), tex2D(texV, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin), tex2D(texV, pixCol, pixLin), tex2D(texV, pixCol + 1, pixLin), tex2D(texV, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin + 1), tex2D(texV, pixCol, pixLin + 1), tex2D(texV, pixCol + 1, pixLin + 1), tex2D(texV, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin + 2), tex2D(texV, pixCol, pixLin + 2), tex2D(texV, pixCol + 1, pixLin + 2), tex2D(texV, pixCol + 2, pixLin + 2)))));
}

// Calculate sinc value
inline __device__ float sincVal(const float dist){
    const float distPi = dist * HIP_PI_F;
    const float distPiP = distPi / 3.f;
    return sinf(distPi) * sinf(distPiP) / (distPi * distPiP);
}

// Calculate coefficient of lanczos interpolation
inline __device__ float lanczosFilter(const float x, const float c0, const float c1, const float c2, const float c3, const float c4, const float c5){
    // Resulting color is the sum of all weighted colors
    float result = 0.f;
    result += c0 * sincVal(x - 2.f);
    result += c1 * sincVal(x - 1.f);
    result += c2 * sincVal(x);
    result += c3 * sincVal(1.f - x);
    result += c4 * sincVal(2.f - x);
    result += c5 * sincVal(3.f - x);
    return result;
}

// Bicubic interpolation for tex y
__global__ void lanczosScaleY(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(255.f * lanczosFilter(distLin,
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin - 2),
            tex2D(texY, pixCol - 1, pixLin - 2),
            tex2D(texY, pixCol, pixLin - 2),
            tex2D(texY, pixCol + 1, pixLin - 2),
            tex2D(texY, pixCol + 2, pixLin - 2),
            tex2D(texY, pixCol + 3, pixLin - 2)),
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin - 1),
            tex2D(texY, pixCol - 1, pixLin - 1),
            tex2D(texY, pixCol, pixLin - 1),
            tex2D(texY, pixCol + 1, pixLin - 1),
            tex2D(texY, pixCol + 2, pixLin - 1),
            tex2D(texY, pixCol + 3, pixLin - 1)),
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin),
            tex2D(texY, pixCol - 1, pixLin),
            tex2D(texY, pixCol, pixLin),
            tex2D(texY, pixCol + 1, pixLin),
            tex2D(texY, pixCol + 2, pixLin),
            tex2D(texY, pixCol + 3, pixLin)),
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin + 1),
            tex2D(texY, pixCol - 1, pixLin + 1),
            tex2D(texY, pixCol, pixLin + 1),
            tex2D(texY, pixCol + 1, pixLin + 1),
            tex2D(texY, pixCol + 2, pixLin + 1),
            tex2D(texY, pixCol + 3, pixLin + 1)),
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin + 2),
            tex2D(texY, pixCol - 1, pixLin + 2),
            tex2D(texY, pixCol, pixLin + 2),
            tex2D(texY, pixCol + 1, pixLin + 2),
            tex2D(texY, pixCol + 2, pixLin + 2),
            tex2D(texY, pixCol + 3, pixLin + 2)),
        lanczosFilter(distCol,
            tex2D(texY, pixCol - 2, pixLin + 3),
            tex2D(texY, pixCol - 1, pixLin + 3),
            tex2D(texY, pixCol, pixLin + 3),
            tex2D(texY, pixCol + 1, pixLin + 3),
            tex2D(texY, pixCol + 2, pixLin + 3),
            tex2D(texY, pixCol + 3, pixLin + 3)))));
}

// Prepares the resample operation
void cuda_resample_aux(AVFrame* src, AVFrame* dst, int operation){
    // Access once
    int srcWidth = src->width, srcHeight = src->height;
    int srcFormat = src->format;
    int dstWidth = dst->width, dstHeight = dst->height;
    int dstFormat = dst->format;

    // Check if is only a format conversion
    bool isOnlyFormatConversion = srcWidth == dstWidth && srcHeight == dstHeight;
    // Changes image pixel format only
    if(isOnlyFormatConversion){
        // Format conversion operation
        omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, dstFormat, dst->data);
        // End resample operation
        return;
    }

    // Get scale ratios
    float scaleHeightRatio = static_cast<float>(dstHeight) / static_cast<float>(srcHeight);
    float scaleWidthRatio = static_cast<float>(dstWidth) / static_cast<float>(srcWidth);

    // Get standard supported pixel format in scaling
    int scaleFormat = getScaleFormat(srcFormat, dstFormat);

    // Calculate the size of the chroma components
    int srcHeightChroma = srcHeight;
    int srcWidthChroma = srcWidth;
    int dstHeightChroma = dstHeight;
    int dstWidthChroma = dstWidth;
    if(scaleFormat == AV_PIX_FMT_YUV422P || scaleFormat == AV_PIX_FMT_YUV420P || scaleFormat == AV_PIX_FMT_YUV422PNORM){
        srcWidthChroma /= 2;
        dstWidthChroma /= 2;
    }
    if(scaleFormat == AV_PIX_FMT_YUV420P){
        srcHeightChroma /= 2;
        dstHeightChroma /= 2;
    }

    // Set configurations of texture memory
    texY.addressMode[0] = hipAddressModeClamp;
    texY.addressMode[1] = hipAddressModeClamp;
    texY.normalized = false;
    texU.addressMode[0] = hipAddressModeClamp;
    texU.addressMode[1] = hipAddressModeClamp;
    texU.normalized = false;
    texV.addressMode[0] = hipAddressModeClamp;
    texV.addressMode[1] = hipAddressModeClamp;
    texV.normalized = false;

    // Temporary buffer
    uint8_t** scaleFormatConverted;
    // Allocate channel buffer pointers
    allocBuffers(scaleFormatConverted, srcWidth, srcHeight, scaleFormat);

    // Format conversion operation
    omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, scaleFormat, scaleFormatConverted);

    // Create channel texture descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

    // Create a 2d cuda array for each component
    hipArray *yArray, *uArray, *vArray;
    hipMallocArray(&yArray, &channelDesc, srcWidth, srcHeight);
    hipMallocArray(&uArray, &channelDesc, srcWidthChroma, srcHeightChroma);
    hipMallocArray(&vArray, &channelDesc, srcWidthChroma, srcHeightChroma);

    // Copy components to cuda arrays
    hipMemcpyToArray(yArray, 0, 0, scaleFormatConverted[0], srcHeight * srcWidth, hipMemcpyHostToDevice);
    hipMemcpyToArray(uArray, 0, 0, scaleFormatConverted[1], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice);
    hipMemcpyToArray(vArray, 0, 0, scaleFormatConverted[2], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice);

    // Bind textures to device memory
    hipBindTextureToArray(&texY, yArray, &channelDesc);
    hipBindTextureToArray(&texU, uArray, &channelDesc);
    hipBindTextureToArray(&texV, vArray, &channelDesc);

    // Set interpolation method
    if(operation == SWS_BILINEAR){
        texY.filterMode = hipFilterModeLinear;
        texU.filterMode = hipFilterModeLinear;
        texV.filterMode = hipFilterModeLinear;
    } else{
        texY.filterMode = hipFilterModePoint;
        texU.filterMode = hipFilterModePoint;
        texV.filterMode = hipFilterModePoint;
    }

    // Free source data
    free2dBuffer(scaleFormatConverted, 3);

    // Create target buffer in device
    uint8_t** scaledDevice;
    int* scaledDeviceSizes;
    // Allocate source buffer in device
    cudaAllocBuffers(scaledDevice, scaledDeviceSizes, dstWidth, dstHeight, scaleFormat);

    // Calculate launch parameters
    pair<dim3, dim3> lumaLP = calculateResizeLP(dstWidth, dstHeight);
    pair<dim3, dim3> chromaLP = calculateResizeLP(dstWidthChroma, dstHeightChroma);

    // Scale each component
    if(operation == SWS_POINT || operation == SWS_BILINEAR){
        scaleTexY << <lumaLP.first, lumaLP.second >> > (srcWidth, srcHeight, dstWidth, dstHeight, scaleWidthRatio, scaleHeightRatio, scaledDevice[0]);
        scaleTexU << <chromaLP.first, chromaLP.second >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice[1]);
        scaleTexV << <chromaLP.first, chromaLP.second >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice[2]);
    } else if(operation == SWS_BICUBIC){
        cubicScaleY << <lumaLP.first, lumaLP.second >> > (srcWidth, srcHeight, dstWidth, dstHeight, scaleWidthRatio, scaleHeightRatio, scaledDevice[0]);
        cubicScaleU << <chromaLP.first, chromaLP.second >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice[1]);
        cubicScaleV << <chromaLP.first, chromaLP.second >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice[2]);
    } else{
        lanczosScaleY << <lumaLP.first, lumaLP.second >> > (srcWidth, srcHeight, dstWidth, dstHeight, scaleWidthRatio, scaleHeightRatio, scaledDevice[0]);
    }

    // Free cuda arrays
    hipFreeArray(yArray);
    hipFreeArray(uArray);
    hipFreeArray(vArray);

    // Temporary buffer
    uint8_t** scaleFormatResized;
    // Allocate channel buffer pointers
    allocBuffers(scaleFormatResized, dstWidth, dstHeightChroma, scaleFormat);

    // Copy resulting data from device
    cudaCopyBuffersFromGPU(scaleFormatResized, scaledDevice, scaledDeviceSizes);

    // Free device memory
    freeCudaMemory(scaledDevice);
    free(scaledDeviceSizes);

    // Format conversion operation
    omp_formatConversion(dstWidth, dstHeight, scaleFormat, scaleFormatResized, dstFormat, dst->data);

    // Free last buffer resources
    free2dBuffer(scaleFormatResized, 3);

    // Sucess
    return;
}

// Wrapper for the cuda resample operation method
int cuda_resample(AVFrame* src, AVFrame* dst, int operation){
    // Access once
    AVPixelFormat srcFormat = static_cast<AVPixelFormat>(src->format);
    AVPixelFormat dstFormat = static_cast<AVPixelFormat>(dst->format);

    // Verify valid frames
    if(src == nullptr || dst == nullptr){
        cerr << "[CUDA] One or both input frames are null!" << endl;
        return -1;
    }

    // Verify valid input data
    if(!src->data || !src->linesize || !dst->data || !dst->linesize){
        cerr << "[CUDA] Frame data buffers can not be null!" << endl;
        return -1;
    }

    // Verify valid input dimensions
    if(src->width < 0 || src->height < 0 || dst->width < 0 || dst->height < 0){
        cerr << "[CUDA] Frame dimensions can not be a negative number!" << endl;
        return -1;
    }

    // Verify if data is aligned
    if(((src->width % 4 != 0 && srcFormat == AV_PIX_FMT_UYVY422) || (dst->width % 4 != 0 && dstFormat == AV_PIX_FMT_UYVY422)) &&
        ((src->width % 12 != 0 && srcFormat == AV_PIX_FMT_V210) || (dst->width % 12 != 0 && dstFormat == AV_PIX_FMT_V210))){
        cerr << "[CUDA] Can not handle unaligned data!" << endl;
        return -1;
    }

    // Verify valid resize
    if((src->width < dst->width && src->height > dst->height) ||
        (src->width > dst->width && src->height < dst->height)){
        cerr << "[CUDA] Can not upscale in an orientation and downscale another!" << endl;
        return -1;
    }

    // Verify if supported conversion
    if(!hasSupportedConversion(srcFormat, dstFormat)){
        cerr << "[CUDA] Pixel format conversion is not supported!" << endl;
        return -1;
    }

    // Verify if supported scaling operation
    if(!isSupportedOperation(operation)){
        cerr << "[CUDA] Scaling operation is not supported" << endl;
        return -1;
    }

    // Variables used
    int duration = -1;
    high_resolution_clock::time_point initTime, stopTime;

    // Start counting operation execution time
    initTime = high_resolution_clock::now();

    // Apply the scaling operation
    cuda_resample_aux(src, dst, operation);

    // Stop counting operation execution time
    stopTime = high_resolution_clock::now();

    // Calculate the execution time
    duration = duration_cast<microseconds>(stopTime - initTime).count();

    // Return execution time of the scaling operation
    return duration;
}