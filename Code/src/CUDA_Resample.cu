#include "hip/hip_runtime.h"
#include "CUDA_Resample.h"

texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texY;
texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texU;
texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat> texV;

// Allocate image channels data buffers depending of the pixel format
void cudaAllocBuffers(uint8_t* &buffer, int* &bufferSize, int width, int height, int pixelFormat){
    // Calculate once
    int wxh = width * height;
    int wxhDiv2 = wxh / 2;
    int wxhDiv4 = wxh / 4;

    // Allocate channel buffer size
    bufferSize = static_cast<int*>(malloc(3 * sizeof(int)));
    // Calculate buffer sizes for each pixel format
    switch(pixelFormat){
        case AV_PIX_FMT_YUV422P:
        case AV_PIX_FMT_YUV422PNORM:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv2;
            bufferSize[2] = wxhDiv2;
            break;
        case AV_PIX_FMT_YUV420P:
            bufferSize[0] = wxh;
            bufferSize[1] = wxhDiv4;
            bufferSize[2] = wxhDiv4;
            break;
    }

    // Allocate buffer memory in device
    hipMalloc((void **) &buffer, bufferSize[0] + bufferSize[1] + bufferSize[2]);
}

// Calculate launch parameters of resize kernel
pair<dim3, dim3> calculateResizeLP(int width, int height){
    // Define the maximum number of thread dim1 size
    int maxNumThreads = 32;

    // Find the thead size
    int vThreadSize = min(maxNumThreads, greatestDivisor(height));
    int hThreadSize = min(maxNumThreads, greatestDivisor(width));

    // Calculate the block size
    int vBlockSize = height / vThreadSize;
    int hBlockSize = width / hThreadSize;

    // Return valid launch parameters
    return pair<dim3, dim3>(dim3(hBlockSize, vBlockSize), dim3(hThreadSize, vThreadSize));
}

// ------------------------------------------------------------------

uint8_t* pinnedHost;
hipChannelFormatDesc channelDesc;
hipArray *ySrc, *uSrc, *vSrc;
uint8_t* scaledDevice;
int* scaledDeviceSizes;
hipStream_t streamY, streamU, streamV;

// Initializes data
void cuda_init(AVFrame* src, AVFrame* dst, int operation){
    // Get standard supported pixel format in scaling
    int scaleFormat = getScaleFormat(src->format, dst->format);

    // Calculate the size of the chroma components
    int srcHeightChroma = src->height;
    int srcWidthChroma = src->width;
    int dstHeightChroma = dst->height;
    int dstWidthChroma = dst->width;
    if(scaleFormat == AV_PIX_FMT_YUV422P || scaleFormat == AV_PIX_FMT_YUV420P || scaleFormat == AV_PIX_FMT_YUV422PNORM){
        srcWidthChroma /= 2;
        dstWidthChroma /= 2;
    }
    if(scaleFormat == AV_PIX_FMT_YUV420P){
        srcHeightChroma /= 2;
        dstHeightChroma /= 2;
    }

    // Allocate host pinned memory
    hipHostMalloc((void **) &pinnedHost, src->height * src->width + 2 * srcHeightChroma * srcWidthChroma + dst->height * dst->width + 2 * dstHeightChroma * dstWidthChroma);

    // Create channel texture descriptor
    channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

    // Set configurations of texture memory
    texY.addressMode[0] = hipAddressModeClamp;
    texY.addressMode[1] = hipAddressModeClamp;
    texY.normalized = false;
    texU.addressMode[0] = hipAddressModeClamp;
    texU.addressMode[1] = hipAddressModeClamp;
    texU.normalized = false;
    texV.addressMode[0] = hipAddressModeClamp;
    texV.addressMode[1] = hipAddressModeClamp;
    texV.normalized = false;

    // Set interpolation method
    if(operation == SWS_BILINEAR){
        texY.filterMode = hipFilterModeLinear;
        texU.filterMode = hipFilterModeLinear;
        texV.filterMode = hipFilterModeLinear;
    } else{
        texY.filterMode = hipFilterModePoint;
        texU.filterMode = hipFilterModePoint;
        texV.filterMode = hipFilterModePoint;
    }

    // Create a 2d cuda array for each source component
    hipMallocArray(&ySrc, &channelDesc, src->width, src->height);
    hipMallocArray(&uSrc, &channelDesc, srcWidthChroma, srcHeightChroma);
    hipMallocArray(&vSrc, &channelDesc, srcWidthChroma, srcHeightChroma);

    // Bind textures to device memory
    hipBindTextureToArray(&texY, ySrc, &channelDesc);
    hipBindTextureToArray(&texU, uSrc, &channelDesc);
    hipBindTextureToArray(&texV, vSrc, &channelDesc);

    // Allocate source buffer in device
    cudaAllocBuffers(scaledDevice, scaledDeviceSizes, dst->width, dst->height, scaleFormat);

    // Create cuda streams for concurrent execution of kernels
    hipStreamCreate(&streamY);
    hipStreamCreate(&streamU);
    hipStreamCreate(&streamV);
}

// Free resources
void cuda_finish(){
    // Delete cuda arrays
    hipFreeArray(ySrc);
    hipFreeArray(uSrc);
    hipFreeArray(vSrc);

    // Free used resources
    hipFree(scaledDevice);
    free(scaledDeviceSizes);

    hipStreamDestroy(streamY);
    hipStreamDestroy(streamU);
    hipStreamDestroy(streamV);

    // Deallocate host pinned memory
    hipHostFree(pinnedHost);
}

// ------------------------------------------------------------------

// Nearest neighbor and bilinear hardware interpolation for tex y
__global__ void scaleTexY(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(tex2D(texY, colOriginal, linOriginal) * 255.f));
}

// Nearest neighbor and bilinear hardware interpolation for tex u
__global__ void scaleTexU(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData, const int dstOffset){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col + dstOffset] = uint8_t(roundf(tex2D(texU, colOriginal, linOriginal) * 255.f));
}

// Nearest neighbor and bilinear hardware interpolation for tex v
__global__ void scaleTexV(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData, const int dstOffset){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col + dstOffset] = uint8_t(roundf(tex2D(texV, colOriginal, linOriginal) * 255.f));
}

// Calculate coefficient of cubic interpolation
inline __device__ float cubicFilter(const float x, const float c0, const float c1, const float c2, const float c3){
    // Resulting color is the sum of all weighted colors
    float result = c0 * (-.6f * x * (x * (x - 2.f) + 1.f));
    result += c1 * (x * x * (1.4f * x - 2.4f) + 1.f);
    result += c2 * (x * (x * (-1.4f * x + 1.8f) + .6f));
    result += c3 * (.6f * x * x * (x - 1.f));
    return result;
}

// Bicubic interpolation for tex y
__global__ void cubicScaleY(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin - 1), tex2D(texY, pixCol, pixLin - 1), tex2D(texY, pixCol + 1, pixLin - 1), tex2D(texY, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin), tex2D(texY, pixCol, pixLin), tex2D(texY, pixCol + 1, pixLin), tex2D(texY, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin + 1), tex2D(texY, pixCol, pixLin + 1), tex2D(texY, pixCol + 1, pixLin + 1), tex2D(texY, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texY, pixCol - 1, pixLin + 2), tex2D(texY, pixCol, pixLin + 2), tex2D(texY, pixCol + 1, pixLin + 2), tex2D(texY, pixCol + 2, pixLin + 2)))));
}

// Bicubic interpolation for tex u
__global__ void cubicScaleU(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData, const int dstOffset){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col + dstOffset] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin - 1), tex2D(texU, pixCol, pixLin - 1), tex2D(texU, pixCol + 1, pixLin - 1), tex2D(texU, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin), tex2D(texU, pixCol, pixLin), tex2D(texU, pixCol + 1, pixLin), tex2D(texU, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin + 1), tex2D(texU, pixCol, pixLin + 1), tex2D(texU, pixCol + 1, pixLin + 1), tex2D(texU, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texU, pixCol - 1, pixLin + 2), tex2D(texU, pixCol, pixLin + 2), tex2D(texU, pixCol + 1, pixLin + 2), tex2D(texU, pixCol + 2, pixLin + 2)))));
}

// Bicubic interpolation for tex v
__global__ void cubicScaleV(const int srcWidth, const int srcHeight, const int dstWidth, const int dstHeight,
    const float scaleWidthRatio, const float scaleHeightRatio, uint8_t* dstData, const int dstOffset){

    // Calculate pixel location
    const int lin = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const int col = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Original index coordinates
    const float linOriginal = ((float) lin + .5f) / scaleHeightRatio - .5f;
    const float colOriginal = ((float) col + .5f) / scaleWidthRatio - .5f;

    // Calculate nearest source sample
    const float pixLin = floorf(linOriginal);
    const float pixCol = floorf(colOriginal);

    // Calculate distance to the source sample
    const float distLin = linOriginal - pixLin;
    const float distCol = colOriginal - pixCol;

    // Assign color
    dstData[__mul24(lin, dstWidth) + col + dstOffset] = uint8_t(roundf(255.f * cubicFilter(distLin,
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin - 1), tex2D(texV, pixCol, pixLin - 1), tex2D(texV, pixCol + 1, pixLin - 1), tex2D(texV, pixCol + 2, pixLin - 1)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin), tex2D(texV, pixCol, pixLin), tex2D(texV, pixCol + 1, pixLin), tex2D(texV, pixCol + 2, pixLin)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin + 1), tex2D(texV, pixCol, pixLin + 1), tex2D(texV, pixCol + 1, pixLin + 1), tex2D(texV, pixCol + 2, pixLin + 1)),
        cubicFilter(distCol, tex2D(texV, pixCol - 1, pixLin + 2), tex2D(texV, pixCol, pixLin + 2), tex2D(texV, pixCol + 1, pixLin + 2), tex2D(texV, pixCol + 2, pixLin + 2)))));
}

// Prepares the resample operation
void cuda_resample_aux(AVFrame* src, AVFrame* dst, int operation,
    int srcWidth, int srcHeight, int dstWidth, int dstHeight,
    int srcWidthChroma, int srcHeightChroma, int dstWidthChroma, int dstHeightChroma,
    int srcFormat, int dstFormat, int scaleFormat,     
    uint8_t* &pinnedHost, hipChannelFormatDesc &channelDesc, hipArray* &ySrc, hipArray* &uSrc, hipArray* &vSrc,
    uint8_t* &scaledDevice, int* &scaledDeviceSizes, hipStream_t &streamY, hipStream_t &streamU, hipStream_t &streamV,
    double* &times){

    // Get scale ratios
    float scaleHeightRatio = static_cast<float>(dstHeight) / static_cast<float>(srcHeight);
    float scaleWidthRatio = static_cast<float>(dstWidth) / static_cast<float>(srcWidth);

    // Buffers for first format conversion
    uint8_t** toScalePtrs = static_cast<uint8_t**>(malloc(3 * sizeof(uint8_t*)));
    toScalePtrs[0] = pinnedHost;
    toScalePtrs[1] = toScalePtrs[0] + srcHeight * srcWidth;
    toScalePtrs[2] = toScalePtrs[1] + srcHeightChroma * srcWidthChroma;
    uint8_t** fromScalePtrs = static_cast<uint8_t**>(malloc(3 * sizeof(uint8_t*)));
    fromScalePtrs[0] = toScalePtrs[2] + srcHeightChroma * srcWidthChroma;
    fromScalePtrs[1] = fromScalePtrs[0] + dstHeight * dstWidth;
    fromScalePtrs[2] = fromScalePtrs[1] + dstHeightChroma * dstWidthChroma;

    // Time variables
    high_resolution_clock::time_point initTime, stopTime;

    // Format conversion operation
    initTime = high_resolution_clock::now();
    omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, scaleFormat, toScalePtrs);
    stopTime = high_resolution_clock::now();
    times[0] = duration_cast<microseconds>(stopTime - initTime).count() * 1.;

    // Calculate launch parameters
    pair<dim3, dim3> lumaLP = calculateResizeLP(dstWidth, dstHeight);
    pair<dim3, dim3> chromaLP = calculateResizeLP(dstWidthChroma, dstHeightChroma);

    // Calculate once the offsets
    int offsetFrom0 = scaledDeviceSizes[0];
    int offsetFrom1 = offsetFrom0 + scaledDeviceSizes[1];

    // Scale each component
    initTime = high_resolution_clock::now();
    if(operation == SWS_POINT || operation == SWS_BILINEAR){
        cudaMemcpyToArrayAsync(ySrc, 0, 0, toScalePtrs[0], srcHeight * srcWidth, hipMemcpyHostToDevice, streamY);
        scaleTexY << <lumaLP.first, lumaLP.second, 0, streamY >> > (srcWidth, srcHeight, dstWidth, dstHeight, scaleWidthRatio, scaleHeightRatio, scaledDevice);
        hipMemcpyAsync(fromScalePtrs[0], scaledDevice, dstHeight * dstWidth, hipMemcpyDeviceToHost, streamY);

        cudaMemcpyToArrayAsync(uSrc, 0, 0, toScalePtrs[1], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice, streamU);
        scaleTexU << <chromaLP.first, chromaLP.second, 0, streamU >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice, offsetFrom0);
        hipMemcpyAsync(fromScalePtrs[1], scaledDevice + offsetFrom0, dstHeightChroma * dstWidthChroma, hipMemcpyDeviceToHost, streamU);

        cudaMemcpyToArrayAsync(vSrc, 0, 0, toScalePtrs[2], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice, streamV);
        scaleTexV << <chromaLP.first, chromaLP.second, 0, streamV >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice, offsetFrom1);
        hipMemcpyAsync(fromScalePtrs[2], scaledDevice + offsetFrom1, dstHeightChroma * dstWidthChroma, hipMemcpyDeviceToHost, streamV);
    } else if(operation == SWS_BICUBIC){
        cudaMemcpyToArrayAsync(ySrc, 0, 0, toScalePtrs[0], srcHeight * srcWidth, hipMemcpyHostToDevice, streamY);
        cubicScaleY << <lumaLP.first, lumaLP.second, 0, streamY >> > (srcWidth, srcHeight, dstWidth, dstHeight, scaleWidthRatio, scaleHeightRatio, scaledDevice);
        hipMemcpyAsync(fromScalePtrs[0], scaledDevice, dstHeight * dstWidth, hipMemcpyDeviceToHost, streamY);

        cudaMemcpyToArrayAsync(uSrc, 0, 0, toScalePtrs[1], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice, streamU);
        cubicScaleU << <chromaLP.first, chromaLP.second, 0, streamU >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice, offsetFrom0);
        hipMemcpyAsync(fromScalePtrs[1], scaledDevice + offsetFrom0, dstHeightChroma * dstWidthChroma, hipMemcpyDeviceToHost, streamU);

        cudaMemcpyToArrayAsync(vSrc, 0, 0, toScalePtrs[2], srcHeightChroma * srcWidthChroma, hipMemcpyHostToDevice, streamV);
        cubicScaleV << <chromaLP.first, chromaLP.second, 0, streamV >> > (srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma, scaleWidthRatio, scaleHeightRatio, scaledDevice, offsetFrom1);
        hipMemcpyAsync(fromScalePtrs[2], scaledDevice + offsetFrom1, dstHeightChroma * dstWidthChroma, hipMemcpyDeviceToHost, streamV);
    }

    // Synchronize device
    hipDeviceSynchronize();
    stopTime = high_resolution_clock::now();
    times[1] = duration_cast<microseconds>(stopTime - initTime).count() * 1.;

    // Free used resources
    free(toScalePtrs);

    // Format conversion operation
    initTime = high_resolution_clock::now();
    omp_formatConversion(dstWidth, dstHeight, scaleFormat, fromScalePtrs, dstFormat, dst->data);
    stopTime = high_resolution_clock::now();
    times[2] = duration_cast<microseconds>(stopTime - initTime).count() * 1.;

    // Free used resources    
    free(fromScalePtrs);

    // Sucess
    return;
}

// Initializes memory if needed and prepares variables
void cuda_resampleStarter(AVFrame* src, AVFrame* dst, int operation,
    int srcWidth, int srcHeight, int dstWidth, int dstHeight, int srcFormat, int dstFormat,
    double* &times){

    // Check if is only a format conversion
    if(srcWidth == dstWidth && srcHeight == dstHeight){
        // Format conversion operation
        omp_formatConversion(srcWidth, srcHeight, srcFormat, src->data, dstFormat, dst->data);
        // End resample operation
        return;
    }

    // Get standard supported pixel format in scaling
    int scaleFormat = getScaleFormat(srcFormat, dstFormat);

    // Calculate the size of the chroma components
    int srcHeightChroma = srcHeight;
    int srcWidthChroma = srcWidth;
    int dstHeightChroma = dstHeight;
    int dstWidthChroma = dstWidth;
    if(scaleFormat == AV_PIX_FMT_YUV422P || scaleFormat == AV_PIX_FMT_YUV420P || scaleFormat == AV_PIX_FMT_YUV422PNORM){
        srcWidthChroma /= 2;
        dstWidthChroma /= 2;
    }
    if(scaleFormat == AV_PIX_FMT_YUV420P){
        srcHeightChroma /= 2;
        dstHeightChroma /= 2;
    }

    // Apply the scaling operation
    cuda_resample_aux(src, dst, operation,
        srcWidth, srcHeight, dstWidth, dstHeight,
        srcWidthChroma, srcHeightChroma, dstWidthChroma, dstHeightChroma,
        srcFormat, dstFormat, scaleFormat,
        pinnedHost, channelDesc, ySrc, uSrc, vSrc,
        scaledDevice, scaledDeviceSizes, streamY, streamU, streamV,
        times);
}

// Wrapper for the cuda resample operation method
int cuda_resample(AVFrame* src, AVFrame* dst, int operation, double* &times){
    // Access once
    int srcWidth = src->width, srcHeight = src->height;
    int srcFormat = src->format;
    int dstWidth = dst->width, dstHeight = dst->height;
    int dstFormat = dst->format;

    // Verify valid frames
    if(src == nullptr || dst == nullptr){
        cerr << "[CUDA] One or both input frames are null!" << endl;
        return -1;
    }

    // Verify valid input data
    if(!src->data || !src->linesize || !dst->data || !dst->linesize){
        cerr << "[CUDA] Frame data buffers can not be null!" << endl;
        return -1;
    }

    // Verify valid input dimensions
    if(srcWidth < 0 || srcHeight < 0 || dstWidth < 0 || dstHeight < 0){
        cerr << "[CUDA] Frame dimensions can not be a negative number!" << endl;
        return -1;
    }

    // Verify if data is aligned
    if(srcFormat == AV_PIX_FMT_UYVY422 || dstFormat == AV_PIX_FMT_UYVY422){
        if((srcWidth % 4 != 0) || (dstWidth % 4 != 0)){
            cerr << "[CUDA] Can not handle unaligned data!" << endl;
            return -1;
        }
        if((srcHeight % 4 != 0) || (dstHeight % 4 != 0)){
            cerr << "[CUDA] Can not handle unaligned data!" << endl;
            return -1;
        }
    }

    if(srcFormat == AV_PIX_FMT_V210 || dstFormat == AV_PIX_FMT_V210){
        if((srcWidth % 12 != 0) || (dstWidth % 12 != 0)){
            cerr << "[CUDA] Can not handle unaligned data!" << endl;
            return -1;
        }
        if((srcHeight % 12 != 0) || (dstHeight % 12 != 0)){
            cerr << "[CUDA] Can not handle unaligned data!" << endl;
            return -1;
        }
    }

    // Verify valid resize
    if((srcWidth < dstWidth && srcHeight > dstHeight) ||
        (srcWidth > dstWidth && srcHeight < dstHeight)){
        cerr << "[CUDA] Can not upscale in an orientation and downscale another!" << endl;
        return -1;
    }

    // Verify if supported conversion
    if(!hasSupportedConversion(srcFormat, dstFormat)){
        cerr << "[CUDA] Pixel format conversion is not supported!" << endl;
        return -1;
    }

    // Verify if supported scaling operation
    if(!isSupportedOperation(operation)){
        cerr << "[CUDA] Scaling operation is not supported" << endl;
        return -1;
    }

    // Apply operation
    cuda_resampleStarter(src, dst, operation, srcWidth, srcHeight, dstWidth, dstHeight, srcFormat, dstFormat, times);

    // Return execution time of the scaling operation
    return 0;
}